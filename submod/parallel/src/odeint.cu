#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdexcept>

#include "odeint.hpp"

namespace gds::parallel::odeint::internal {

__global__ void odeint_kernel(double* result, double* x, double* y, double n) {
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tidx < n) {
    double x0 = x[tidx];
    double x1 = x[tidx + 1];
    double y0 = y[tidx];
    double y1 = y[tidx + 1];

    result[tidx] = (y0 + y1) * (x1 - x0) * 0.5;
  }
}

};  // namespace gds::parallel::odeint::internal

namespace gds::parallel::odeint {

double integrate(std::vector<double>& x, std::vector<double>& y) {
  // check input size
  if (x.size() != y.size()) {
    throw std::invalid_argument("x and y must have the same size");
  }

  // allocate device memory
  double* x_dev;
  double* y_dev;
  hipMalloc(&x_dev, x.size() * sizeof(double));
  hipMalloc(&y_dev, y.size() * sizeof(double));

  // copy data to device
  hipMemcpy(x_dev, x.data(), x.size() * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(y_dev, y.data(), y.size() * sizeof(double), hipMemcpyHostToDevice);

  // config thread block and grid size
  int n          = x.size();
  int block_size = 256;
  int grid_size  = (n + block_size - 1) / block_size;

  // allocate device memory for result
  double* result_dev;
  hipMalloc(&result_dev, n * sizeof(double));

  // call kernel
  gds::parallel::odeint::internal::odeint_kernel<<<grid_size, block_size>>>(result_dev, x_dev, y_dev, n);

  // copy result back to host
  double* result = (double*)malloc(n * sizeof(double));
  hipMemcpy(result, result_dev, n * sizeof(double), hipMemcpyDeviceToHost);

  // epilogue
  double integral = 0.0;
  for (int i = 0; i < n - 1; i++) {
    integral += result[i];
  }

  // free device memory
  hipFree(x_dev);
  hipFree(y_dev);
  hipFree(result_dev);
  delete[] result;

  return integral;
}

}  // namespace gds::parallel::odeint
